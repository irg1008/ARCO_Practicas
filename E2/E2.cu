/*
* ARQUITECTURA DE COMPUTADORES
* 2� Grado en Ingenieria Informatica
*
* PRACTICA 2: "Ordenaci�n de Array De Menor a Mayor".
* >> TODO => Finalizado.
*
* AUTOR: Iv�n Ruiz G�zquez e Iv�n Maeso Adri�n.
*/
///////////////////////////////////////////////////////////////////////////
// Includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>


// Defines
#define RAN_MIN 1
#define RAN_MAX 50

// Declaracion de funciones
void cudaDev()
{
	// Saca num hilos, funcion CUDA
	int dev;
	hipGetDevice(&dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	//
	printf("\n***********************************************************************\n\n");
	printf("> Nombre Dispositivos: %s\n", deviceProp.name);
	printf("> Capacidad de Computo: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("> Numero de MultiProcesadores: %d \n", deviceProp.multiProcessorCount);
	printf("> Numero de Nucleos (Arq. PASCAL): %d \n", 64);
	printf("> Maximo de hilos por eje en bloque\n");
	printf(" \t[x -> %d]\n \t[y -> %d]\n \t[z -> %d]\n",deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("> Maximo de bloques por eje\n");
	printf(" \t[x -> %d]\n \t[y -> %d]\n \t[z -> %d]\n",deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("\n***********************************************************************\n");
}

__global__
void ordenarArray(int *dev_desordenado, int *dev_ordenado, int elem)
{
	int myID = threadIdx.x;
	int rango = 0;

	for(int i=0; i<elem; i++) {
		if((dev_desordenado[myID] > dev_desordenado[i]) && (myID != i))
			rango++;
		if(dev_desordenado[myID] == dev_desordenado[i] && myID > i)
			rango++;
	}

	dev_ordenado[rango] = dev_desordenado[myID];
}

// MAIN: Rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	//Eventos
	hipEvent_t start;
	hipEvent_t stop;

	// Creacion de eventos
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Marca de inicio
	hipEventRecord(start, 0);

	// Declaracion
	int *hst_desordenado;
	int *hst_ordenado;
	int *dev_desordenado;
	int *dev_ordenado;

	// Elementos
	int elem;

	// Llama a la funci�n Cuda que devuelve info
	cudaDev();

	// Pregunta n�mero de elemetos
	do {
		printf("\n\nNumero de elementos (MAX=1024): ");
		scanf("%d", &elem);
		getchar();
	} while (elem<=0 || elem>1024);

	// Dimensiones del kernel
	dim3 Nbloques(1);
	dim3 hilosB(elem);

	// Reserva en el host
	hst_ordenado = (int*)malloc(elem * sizeof(int));
	hst_desordenado = (int*)malloc(elem * sizeof(int));

	// Reserva en el device
	hipMalloc( &dev_ordenado, elem * sizeof(int));
	hipMalloc( &dev_desordenado, elem * sizeof(int));

	
	// Insertamos valores random en la matriz
	srand((int)time(NULL));
	for (int i = 0; i < elem; i++)
	{
		hst_desordenado[i] = RAN_MIN + rand() % RAN_MAX;
	}

	// Pasamos el array al device y le damos la vuelta
	hipMemcpy(dev_desordenado, hst_desordenado, elem * sizeof(int), hipMemcpyHostToDevice);
	ordenarArray <<<Nbloques, hilosB>>>(dev_desordenado, dev_ordenado, elem);

	// Check de errores
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	// Pasamos el resultado a la cpu
	hipMemcpy(hst_ordenado, dev_ordenado, elem * sizeof(int), hipMemcpyDeviceToHost);

	// Muestra contenido de arrays y resultado
	printf("\n\nMatriz Desordenada: \n");
	printf("*********************\n");
	for (int i = 0; i < elem; i++) {
		printf("%d ", hst_desordenado[i]);
	}

	printf("\n\nMatriz Ordenada: \n");
	printf("*********************\n");
	for (int i = 0; i < elem; i++) {
		printf("%d ", hst_ordenado[i]);
	}
	
	// Marca de final
	hipEventRecord(stop, 0);

	// Sincronizacion CPU-GPU
	hipEventSynchronize(stop);

	// Calculo del tiempo
	float tiempoTrans;
	hipEventElapsedTime(&tiempoTrans, start, stop);
	printf("\n\n\n> Tiempo de ejecuccion: %f ms\n", tiempoTrans);

	// Liberacion de recursos
	free(hst_desordenado);
	free(hst_ordenado);
	hipFree(dev_desordenado);
	hipFree(dev_ordenado);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Salida
	time_t fecha;
	time(&fecha);
	printf("\n\n***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}
