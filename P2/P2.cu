﻿// includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define N 8
// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	// declaracion
	float *hst_matriz;
	float *dev_matriz;
	float *hst1_matriz;
	float *dev1_matriz;
	// reserva en el host
	hst_matriz = (float*)malloc( N*sizeof(float) );
	hst1_matriz = (float*)malloc( N*sizeof(float) );
	// reserva en el device
	hipMalloc( (void**)&dev_matriz, N*sizeof(float) );
	hipMalloc( (void**)&dev1_matriz, N*sizeof(float) );
	// inicializacion de datos en el host
	srand ( (int)time(NULL) );
	for (int i=0; i<N; i++)
	{
		hst_matriz[i] = (float) rand() / RAND_MAX;
	}
	// visualizacion de datos en el host
	printf("DATOS:\n");
	for (int i=0; i<N; i++)
	{
		printf("A[%i] = %.2f\n", i, hst_matriz[i]);
	}
	// copia de datos CPU -> GPU
	hipMemcpy(dev_matriz, hst_matriz, N*sizeof(float), hipMemcpyHostToDevice);
	// copia de datos GPU -> GPU
	hipMemcpy(dev1_matriz, dev_matriz, N*sizeof(float), hipMemcpyDeviceToDevice);
	// copia de datos GPU -> CPU
	hipMemcpy(hst1_matriz, dev1_matriz, N*sizeof(float), hipMemcpyDeviceToHost);
	// visualizacion de datos en el host
	printf("\nDATOS:\n");
	for (int i=0; i<N; i++)
	{
		printf("A[%i] = %.2f\n", i, hst1_matriz[i]);
	}
	// salida
	time_t fecha;
	time(&fecha);
	printf("***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}