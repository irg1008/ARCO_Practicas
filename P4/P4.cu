﻿/*
* ARQUITECTURA DE COMPUTADORES
* 2º Grado en Ingenieria Informatica
*
* PRACTICA 2: "Suma De Matrices Paralela"
* >> Arreglar for en __global__
* >> Pasar numElem como argumento
*
* AUTOR: Ivanes
*/
///////////////////////////////////////////////////////////////////////////
// Includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


// Defines
#define RAN_MIN 1
#define RAN_MAX 9
// Bloques - Hilos
#define MAX_HILOS 10
#define MAX_BLOQUES 512

// Declaracion de funciones
int numHilos()
{
	int numHilos;

	// Saca num hilos, funcion CUDA
	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	// deviceProp.maxThreadsPerBlock;
	// deviceProp.maxGridSize[0];
	int maxValores = MAX_HILOS*MAX_BLOQUES;


	//
	printf("\n***********************************************************************\n\n");
	printf("> Nombre Dispositivos: %s\n", deviceProp.name);
	printf("> Capacidad de Computo: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("> Numero de MultiProcesadores: %d \n", deviceProp.multiProcessorCount);
	printf("> Numero de Nucleos (Arq. PASCAL): %d \n", 64);
	printf("> Maximo de hilos por eje en bloque\n");
	printf(" \t[x -> %d]\n \t[y -> %d]\n \t[z -> %d]\n",deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("> Maximo de bloques por eje\n");
	printf(" \t[x -> %d]\n \t[y -> %d]\n \t[z -> %d]\n",deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("\n***********************************************************************\n");

	printf("\nEl numero maximo de elementos del array es: %d valores\n", maxValores);
	do {
		printf("\n\nCuantos elementos quieres que tenga los vectores: ");
		scanf("%d", &numHilos);
		getchar();
	} while ((numHilos > maxValores) || (numHilos <= 0));

	return numHilos;
}

__global__
void reverseMatriz(int *dev_matriz, int *dev_matriz_reverse, int *dev_matriz_resultado, int numElem)
{
	// Crea la matriz inversa
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	dev_matriz_reverse[id] = dev_matriz[numElem - 1 - id];

	// Suma las matrices
	dev_matriz_resultado[id] = dev_matriz[id] + dev_matriz_reverse[id];
}

// MAIN: Rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	// Declaracion
	int *hst_matriz;
	int *hst_matriz_reverse;
	int *hst_matriz_resultado;
	int *dev_matriz;
	int *dev_matriz_reverse;
	int *dev_matriz_resultado;

	// Saca numero de hilos y pregunta cuantos elementos quiere en el array. Pone el número de bloques a usar, 1 en este caso
	int numElem = numHilos();
	int numBlock = numElem/MAX_HILOS;
	
	// Obtencion del numero del bloques
	if(numElem%MAX_HILOS != 0)
		numBlock++;
	printf("Lanzamos %d valores en %d bloques de %d hilos", numElem, numBlock, MAX_HILOS);


	// Reserva en el host
	hst_matriz = (int*)malloc(numElem * sizeof(int));
	hst_matriz_reverse = (int*)malloc(numElem * sizeof(int));
	hst_matriz_resultado = (int*)malloc(numElem * sizeof(int));

	// Reserva en el device
	hipMalloc( &dev_matriz, numElem * sizeof(int));
	hipMalloc( &dev_matriz_reverse, numElem * sizeof(int));
	hipMalloc( &dev_matriz_resultado, numElem * sizeof(int));

	// Insertamos valores random en la matriz
	srand((int)time(NULL));
	for (int i = 0; i < numElem; i++)
	{
		hst_matriz[i] = RAN_MIN + rand() % RAN_MAX;
	}

	// Pasamos el array al device y le damos la vuelta
	hipMemcpy(dev_matriz, hst_matriz, numElem * sizeof(int), hipMemcpyHostToDevice);
	reverseMatriz <<< numBlock, MAX_HILOS>>>(dev_matriz, dev_matriz_reverse, dev_matriz_resultado, numElem);

	// Check de errores
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	// Pasamos el array inverso a la cpu
	hipMemcpy(hst_matriz_reverse, dev_matriz_reverse, numElem * sizeof(int), hipMemcpyDeviceToHost);

	// Pasamos el resultado a la cpu
	hipMemcpy(hst_matriz_resultado, dev_matriz_resultado, numElem * sizeof(int), hipMemcpyDeviceToHost);

	// Muestra contenido de arrays y resultado
	printf("\n\nMatriz: \n");
	for (int i = 0; i < numElem; i++)
		printf("%d ", hst_matriz[i]);

	printf("\n\nMatriz Inversa: \n");
	for (int i = 0; i < numElem; i++)
		printf("%d ", hst_matriz_reverse[i]);

	printf("\n\nMatriz Resultado: \n");
	for (int i = 0; i < numElem; i++)
		printf("%d ", hst_matriz_resultado[i]);

	free(hst_matriz);
	free(hst_matriz_reverse);
	free(hst_matriz_resultado);

	hipFree(dev_matriz);
	hipFree(dev_matriz_reverse);
	hipFree(dev_matriz_resultado);

	// salida
	time_t fecha;
	time(&fecha);
	printf("\n\n***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}
