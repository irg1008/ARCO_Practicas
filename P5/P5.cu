#include "hip/hip_runtime.h"
/*
* ARQUITECTURA DE COMPUTADORES
* 2� Grado en Ingenieria Informatica
*
* PRACTICA 2: "Suma De Matrices Paralela"
* >> Arreglar for en __global__
* >> Pasar numElem como argumento
*
* AUTOR: Ivanes
*/
///////////////////////////////////////////////////////////////////////////
// Includes
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include ""

// Defines
#define RAN_MIN 1
#define RAN_MAX 9

// Declaracion de funciones
void cudaDev()
{
	// Saca num hilos, funcion CUDA
	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	//
	printf("\n***********************************************************************\n\n");
	printf("> Nombre Dispositivos: %s\n", deviceProp.name);
	printf("> Capacidad de Computo: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("> Numero de MultiProcesadores: %d \n", deviceProp.multiProcessorCount);
	printf("> Numero de Nucleos (Arq. PASCAL): %d \n", 64);
	printf("> Maximo de hilos por eje en bloque\n");
	printf(" \t[x -> %d]\n \t[y -> %d]\n \t[z -> %d]\n",deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
	printf("> Maximo de bloques por eje\n");
	printf(" \t[x -> %d]\n \t[y -> %d]\n \t[z -> %d]\n",deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
	printf("\n***********************************************************************\n");
}

__global__
void binaryConv(int *dev_matriz, int *dev_matriz_resultado)
{
	// Crea la matriz inversa
	int columna = threadIdx.x;
	int fila = threadIdx.y;

	int myID = fila + columna * blockDim.x;

	// Convierte la matriz (<5 -> 0; >=5 -> 1)
	if(dev_matriz[myID] < 5)
		dev_matriz_resultado[myID] = 0;
	else
		dev_matriz_resultado[myID] = 1;

}

// MAIN: Rutina principal ejecutada en el host
int main(int argc, char** argv)
{
	// Declaracion
	int *hst_matriz;
	int *hst_matriz_resultado;
	int *dev_matriz;
	int *dev_matriz_resultado;

	// Filas y Columnas
	int filas;
	int columnas;

	// Llama a la funci�n Cuda que devuelve info
	cudaDev();

	// Pregunta n�mero de filas y columnas
	printf("Numero maximo de elementos: 1024");
	do {
		printf("\n\nNumero de filas: ");
		scanf("%d", &filas);
		getchar();

		printf("\nNumero de columnas: ");
		scanf("%d", &columnas);
		getchar();
	} while ((filas*columnas > 1024) || filas <= 0 || columnas <= 0);

	// Saca el tama�o del array
	printf("\nNumero de elementos: %d", filas*columnas);

	// Dimensiones del kernel
	dim3 Nbloques(1);
	dim3 hilosB(columnas, filas);

	// Reserva en el host
	hst_matriz = (int*)malloc(filas*columnas * sizeof(int));
	hst_matriz_resultado = (int*)malloc(filas*columnas * sizeof(int));

	// Reserva en el device
	hipMalloc( &dev_matriz, filas*columnas * sizeof(int));
	hipMalloc( &dev_matriz_resultado, filas*columnas * sizeof(int));

	// Insertamos valores random en la matriz
	srand((int)time(NULL));
	for (int i = 0; i < filas*columnas; i++)
	{
		hst_matriz[i] = RAN_MIN + rand() % RAN_MAX;
	}

	// Pasamos el array al device y le damos la vuelta
	hipMemcpy(dev_matriz, hst_matriz, filas*columnas * sizeof(int), hipMemcpyHostToDevice);
	binaryConv <<<Nbloques, hilosB>>>(dev_matriz, dev_matriz_resultado);

	// Check de errores
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
		exit(-1);
	}

	// Pasamos el resultado a la cpu
	hipMemcpy(hst_matriz_resultado, dev_matriz_resultado, filas*columnas * sizeof(int), hipMemcpyDeviceToHost);

	// Muestra contenido de arrays y resultado
	printf("\n\nMatriz: \n");
	printf("*****************\n");

	for (int i = 0; i < filas; i++) {
		for(int j = 0; j < columnas, j++) {
			printf("%d ", hst_matriz[i+j*columnas]);
		}
		printf("\n");
	}

	printf("\n\nMatriz Resultado: \n");
	printf("*********************\n");

	for (int i = 0; i < filas; i++) {
		for(int j = 0; j < columnas, j++) {
			printf("%d ", hst_matriz_resultado[i+j*columnas]);
		}
		printf("\n");
	}
	

	free(hst_matriz);
	free(hst_matriz_resultado);

	hipFree(dev_matriz);
	hipFree(dev_matriz_resultado);

	// salida
	time_t fecha;
	time(&fecha);
	printf("\n\n***************************************************\n");
	printf("Programa ejecutado el: %s\n", ctime(&fecha));
	printf("<pulsa [INTRO] para finalizar>");
	getchar();
	return 0;
}
